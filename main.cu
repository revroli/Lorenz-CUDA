#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <fstream>

#include "DeviceProperties.cuh"

#define RK_ORDER 4
#define BUTCHER_SIZE RK_ORDER + 1

using namespace std;

__device__ void Lorenz(float*, float*, float);
__global__ void RungeKutta4(float*, float*, int);

void Linspace(float*, float, float, int);

__constant__ float const_d_A[(RK_ORDER-1)*(RK_ORDER-1)];
__constant__ float const_d_B[RK_ORDER];

int main()
{

	int Resolution = 1536*64*100; // Threads per SM * number of SMs * 3
	int BlockSize  = 128;
	int GridSize = Resolution/BlockSize + (Resolution % BlockSize == 0 ? 0:1);
	
	ListCUDADevices();
	
	int MajorRevision  = 8;
	int MinorRevision  = 6;
	int SelectedDevice = SelectDeviceByClosestRevision(MajorRevision, MinorRevision);
	
	PrintPropertiesOfSpecificDevice(SelectedDevice);
	hipSetDevice(SelectedDevice);
	
	float* h_State      = (float*)aligned_alloc(64, 3*Resolution * sizeof(float));
	float* h_Parameters = (float*)aligned_alloc(64,   Resolution * sizeof(float));
	float* h_A = (float*)aligned_alloc(64,   (RK_ORDER - 1) * (RK_ORDER - 1) * sizeof(float));
	float* h_B = (float*)aligned_alloc(64,   RK_ORDER * sizeof(float));

	float* d_State;
	float* d_Parameters;
	hipMalloc((void**)&d_State,      3*Resolution * sizeof(float));
	hipMalloc((void**)&d_Parameters,   Resolution * sizeof(float));

	// Initialisation
	Linspace(h_Parameters, 0.0, 40.0, Resolution);
	for (int i = 1; i<Resolution; i++)
	{
		h_State[i]              = -5.0;
		h_State[i+Resolution]   =  0.0;
		h_State[i+2*Resolution] =  0.0;
	}	

	h_A[0] = 0.5f; h_A[1] = 0.0f; h_A[2] = 0.0f;
	h_A[3] = 0.0f; h_A[4] = 0.5f; h_A[5] = 0.0f;
	h_A[6] = 0.0f; h_A[7] = 0.0f; h_A[8] = 1.0f;

	h_B[0] = 1.0f/6.0f;
	h_B[1] = 1.0f/3.0f;
	h_B[2] = 1.0f/3.0f;
	h_B[3] = 1.0f/6.0f;


	hipMemcpy(d_State, h_State, 3*sizeof(float)*Resolution, hipMemcpyHostToDevice);
	hipMemcpy(d_Parameters, h_Parameters, sizeof(float)*Resolution, hipMemcpyHostToDevice);
	
	//Butcher Tableau to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(const_d_A), h_A, (RK_ORDER - 1) * (RK_ORDER - 1) * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(const_d_B), h_B, sizeof(float) * RK_ORDER);

	//Kernel run
	RungeKutta4<<<GridSize, BlockSize>>> (d_State, d_Parameters, Resolution);
	hipDeviceSynchronize();
	
	//Save the products
	hipMemcpy(h_State, d_State, 3*sizeof(float)*Resolution, hipMemcpyDeviceToHost);
	

	ofstream outfile("output_files/main_output.txt");
	outfile << std::setprecision(8) << std::fixed;
	outfile << "# Lorenz System CUDA Simulation Output\n";
	outfile << "# Columns: Parameter X Y Z\n";
	outfile << "# Resolution: " << Resolution << "\n";
	outfile << "# Each row: <parameter> <X> <Y> <Z>\n";
	for (int i = 0; i < Resolution; ++i) {
		outfile << h_Parameters[i] << " "
				<< h_State[i] << " "
				<< h_State[i + Resolution] << " "
				<< h_State[i + 2 * Resolution] << "\n";
	}
	outfile.close();
}

__forceinline__ __device__ void Lorenz(float* F, float* X, float P)
{
	F[0] = float(10)*(X[1] - X[0]);
	F[1] = P*X[0] - X[1] - X[0]*X[2];
	F[2] = X[0]*X[1] - float(2.666) * X[2];
}

__global__ void RungeKutta4(float* d_State, float* d_Parameters, int N)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (tid < N)
	{
		float X[3] = {d_State[tid], d_State[tid+N], d_State[tid+2*N]};

		float P = d_Parameters[tid];
		
		// van egy k vector
		//implicitet nem lehet így kiszámolni, úgyhogy csak az explicitet számoljuk

		float k[RK_ORDER * 3];		//hogyan rendezem? legyen [iteráció][x-dimenzió]
		float x[3];
		float intersum;
		
		float T = 0;
		float h = 0.001; //DT

		int i_minus;
		
		for (int n=0; n<10000; n++)
		{
			Lorenz(k, X, P);		//kn1

			#pragma unroll
			for (int i = 1; i < RK_ORDER; i++){
				
				i_minus = i-1;

				#pragma unroll
				for (int k_iter = 0; k_iter < 3; k_iter++){

					intersum = 0;

					#pragma unroll
					for (int j=0; j < i; j++){
						intersum += k[j*3 + k_iter] * const_d_A[(i_minus) * 3 + j];	//a a 00-ból kell induljon 
																//ezt átírni valahogy 1 MA-ra?
																// unrollal biztos kijön
																//(i-1)*-at elég lehet csak 1-szer kiszámolni
					}
					
					x[k_iter] = X[k_iter] + h  * intersum;
				}	
				
				Lorenz(k + 3*i, x, P);
			}

			#pragma unroll
			for (int i = 0; i < 3; i++){
				intersum = 0;

				#pragma unroll
				for (int j = 0; j < RK_ORDER; j++){
					intersum += const_d_B[j] * k[3*j + i]; 
				}
				
				X[i] = X[i] + h * intersum;
			}

			T += h; //kihagyható amúgy
		}
		
		d_State[tid] = X[0];
		d_State[tid + N] = X[1];
		d_State[tid + 2*N] = X[2];
	}
}

void Linspace(float* x, float B, float E, int N)
{
    float Increment;
	
	x[0] = B;
	
	if ( N>1 )
	{
		x[N-1] = E;
		Increment = (E-B)/(N-1);
		
		for (int i=1; i<N-1; i++)
		{
			x[i] = B + i*Increment;
		}
	}
}